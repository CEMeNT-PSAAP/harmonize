struct _24b8 { unsigned long long int data[3]; };
struct _1048584b8 { unsigned long long int data[131073]; };
struct _0b8 { unsigned long long int data[0]; };
extern "C" __device__ int _initialize(void*, void* prog);
extern "C" __device__ int _finalize  (void*, void* prog);
extern "C" __device__ int _make_work (bool* result, void* prog);
extern "C" __device__ int _odd(void*, void* fn_param_1, void* fn_param_2);
extern "C" __device__ int _even(void*, void* fn_param_1, void* fn_param_2);
struct Odd;
struct Even;
struct Odd {
	using Type = void(*)(_24b8);
	template<typename PROGRAM>
	__device__ static void eval(PROGRAM prog, _24b8 fn_param_2) {
		int  dummy_void_result = 0;
		int *fn_param_0 = &dummy_void_result;
		_odd(fn_param_0, &prog, &fn_param_2);
	}
};
struct Even {
	using Type = void(*)(_24b8);
	template<typename PROGRAM>
	__device__ static void eval(PROGRAM prog, _24b8 fn_param_2) {
		int  dummy_void_result = 0;
		int *fn_param_0 = &dummy_void_result;
		_even(fn_param_0, &prog, &fn_param_2);
	}
};
struct collaz{
	static const size_t STASH_SIZE = 8;
	static const size_t FRAME_SIZE = 8192;
	static const size_t POOL_SIZE = 8192;
	typedef OpUnion<Odd,Even> OpSet;
	typedef _1048584b8* DeviceState;
	typedef _0b8* GroupState;
	typedef _0b8* ThreadState;
	template<typename PROGRAM>
	__device__ static void initialize(PROGRAM prog) {
		int  dummy_void_result = 0;
		int *fn_param_0 = &dummy_void_result;
		_initialize(fn_param_0, &prog);
	}
	template<typename PROGRAM>
	__device__ static void finalize(PROGRAM prog) {
		int  dummy_void_result = 0;
		int *fn_param_0 = &dummy_void_result;
		_finalize(fn_param_0, &prog);
	}
	template<typename PROGRAM>
	__device__ static bool make_work(PROGRAM prog) {
		bool  result;
		bool *fn_param_0 = &result;
		_make_work(fn_param_0, &prog);
		return result;
	}
};
