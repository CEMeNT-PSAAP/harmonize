
#include <hip/hip_runtime.h>
#include "math.h"
#include <vector>





namespace util {




template<typename T>
struct PairEquivalent;

template<>
struct PairEquivalent<unsigned short>
{
	typedef unsigned int Type;
};


template<>
struct PairEquivalent<unsigned int>
{
	typedef unsigned long long int Type;
};


template<typename T>
struct PairPack
{

	typedef PairPack<T> Self;

	typedef typename PairEquivalent<T>::Type PairType;

	static const PairType RIGHT_MASK = std::numeric_limits<T>::max();
	static const size_t   HALF_WIDTH = std::numeric_limits<T>::digits;
	static const PairType LEFT_MASK  = RIGHT_MASK << HALF_WIDTH;

	PairType data;

	__host__  __device__ T    get_left() {
		return (data >> HALF_WIDTH) & RIGHT_MASK;
	}

	__host__  __device__ void set_left(T val) {
		data &= RIGHT_MASK;
		data |= ((PairType) val) << HALF_WIDTH;
	}

	__host__  __device__ T    get_right(){
		return data & RIGHT_MASK;
	}

	__host__  __device__ void set_right(T val){
		data &= LEFT_MASK;
		data |= val;
	}

	PairPack<T> () = default;


	__host__  __device__ PairPack<T> (T left, T right){
		data   = left;
		data <<= HALF_WIDTH;
		data  |= right;
	}

};



template<typename T>
struct DevObj
{
	T* adr;

	__host__ DevObj<T>(){
		hipMalloc( (void**) &adr,  sizeof(T)  );
	}

	__host__ DevObj<T>(size_t size){
		hipMalloc( (void**) &adr,  sizeof(T)*size  );
	}

	__host__ ~DevObj<T>(){
		hipFree(adr);
	}

};


template<typename T>
struct DevVec
{
	size_t size;
	T* adr;

	__host__ DevVec<T>(size_t s){
		size = s;
		hipMalloc( (void**) &adr,  sizeof(T)*size  );
	}

	__host__ ~DevVec<T>(){
		hipFree(adr);
	}

	__host__ void operator<<(std::vector<T> &other) {
		hipMemcpy(adr,other.data(),sizeof(T)*size,hipMemcpyHostToDevice);
	}

	__host__ void operator>>(std::vector<T> &other) {
		hipMemcpy(other.data(),adr,sizeof(T)*size,hipMemcpyDeviceToHost);
	}

};


/*
// Gives a count of how many threads in the current warp with a lower warp-local id are currently
// active. This is useful for finding an index for a set of operations performed once by an
// unpredictable number of threads within a warp.
*/
 __device__ unsigned int warp_inc_scan(){

	unsigned int active = __activemask();
	unsigned int keep = (1 << threadIdx.x) - 1;
	unsigned int scan = __popc(active & keep);
	return scan;

}


/*
// This function returns the number of currently active threads in a warp
*/
 __device__ unsigned int active_count(){
	return __popc(__activemask()); 
}


/*
// This returns true only if the current thread is the active thread with the lowest warp-local id.
// This is valuable for electing a "leader" to perform single-threaded work for a warp.
*/
 __device__ bool current_leader(){
	return ((__ffs(__activemask())-1) == threadIdx.x);
}


/*
// A simple pseudo-random number generator. This algorithm should never be used for cryptography, 
// it is simply used to generate numbers random enough to reduce collisions for atomic
// instructions performed to manage the runtime state.
*/
 __device__ unsigned int random_uint(unsigned int &state){

	state = (0x10DCDu * state + 1u);
	return state;

}



template<typename ITER_TYPE>
struct BasicIter
{


	typedef ITER_TYPE IterType;

	IterType value;
	IterType limit;
	IterType width;


	 __device__ bool step(IterType& iter_val){

		if( value >= limit ){
			return false;
		}

		iter_val = value;
		value += width;
		return true;

	}

};



template<typename ITER_TYPE>
struct GroupWorkIter
{


	typedef ITER_TYPE IterType;

	IterType start;
	IterType limit;
	IterType chunk;
	IterType chunk_limit;

	 __device__ void reset(IterType start_val, IterType limit_val) {
		__syncwarp();
		if( current_leader() ){
			start = start_val;
			limit = limit_val;
			chunk = 0;
			IterType iter_width = limit_val - start_val;
			chunk_limit = iter_width / blockDim.x;
			if( (iter_width % blockDim.x) != 0 ){
				chunk_limit += 1;
			}
		}
		__syncwarp();
	}


	 __device__ bool step(IterType& iter_val) {
		
		if( chunk <= chunk_limit ){
			IterType val = start + chunk*blockDim.x + threadIdx.x;	
			if ( val < limit ){
				iter_val = val;
			}
			__syncwarp();
			if( current_leader() ){
				chunk += 1;
			}
			__syncwarp();
			return val < limit;
		}
		return false;
		
	}


	template<size_t MULTIPLIER>
	 __device__ BasicIter<IterType> multi_step() {
	
		BasicIter<IterType> result;
		result.value = 0;
		result.limit = 0;
		result.width = blockDim.x;
		if( chunk <= chunk_limit ){
			IterType start_val = start + chunk*blockDim.x + threadIdx.x;	
			IterType limit_val = start_val + MULTIPLIER*blockDim.x;
			if ( start_val < limit ){
				result.value = start_val;
			} else {
				result.value = limit;
			}
			if ( limit_val < limit ){
				result.limit = limit_val;
			} else {
				result.limit = limit;
			}
			__syncwarp();
			if( current_leader() ){
				chunk += MULTIPLIER;
			}
			__syncwarp();
		}
		return result;
		
	}


	 __device__ bool done() {
		
		return ( chunk > chunk_limit );
		
	}


};



struct GlobalTurnstile
{

	unsigned long long int counter;

	__host__ static void reset(GlobalTurnstile* turnstile) {
		
	}


	 __device__ bool cross() {
	
		unsigned long long int checkout_index = atomicAdd(&counter,1);

		if(checkout_index == (gridDim.x-1)){
			atomicExch(&counter,0); 
			return true;
		} else {
			return false;
		}
		
	}

};



void check_error(){

	hipError_t status = hipGetLastError();

	if(status != hipSuccess){
		const char* err_str = hipGetErrorString(status);
		printf("ERROR: \"%s\"\n",err_str);
	}

}





void cli_graph(float* data, int size, int width, int height, float low, float high){

	const char* lookup[25] = {
		"⠀","⡀","⡄","⡆","⡇",
		"⢀","⣀","⣄","⣆","⣇",
		"⢠","⣠","⣤","⣦","⣧",
		"⢰","⣰","⣴","⣶","⣷",
		"⢸","⣸","⣼","⣾","⣿"
	};

	
	float max = 0;
	for( int i=0; i<size; i++){
		if( data[i] > max ){
			max = data[i];
		}
	}

	printf("Max is %f\n",max);

	int x_iter;
	float l_val, r_val;
	float last=0;

	printf("%7.5f_\n",max);
	for(int i=0; i<height; i++){
		float base = (height-i-1)*max/height;
		printf("%7.5f_",base);
		x_iter = 0;
		for(int j=0; j<width; j++){
			l_val = 0;
			r_val = 0;
			int l_limit = (j*2*size)/(width*2);
			int r_limit = ((j*2+1)*size)/(width*2);
			float count = 0.0;
			for(; x_iter < l_limit; x_iter++){
				l_val += data[x_iter];
				//printf("%f,",data[x_iter]);
				count += 1.0;
			}
			l_val = ( count == 0.0 ) ? last : l_val / count;
			last = l_val;
			count = 0.0;
			for(; x_iter < r_limit; x_iter++){
				r_val += data[x_iter];
				count += 1.0;
			}
			r_val = ( count == 0.0 ) ? last : r_val / count;
			last = r_val;
			l_val = ( l_val - base )/max*height*4;
			r_val = ( r_val - base )/max*height*4;
			int l_idx = (l_val <= 0.0) ? 0 : ( (l_val >= 4.0) ? 4 : l_val );
			int r_idx = (r_val <= 0.0) ? 0 : ( (r_val >= 4.0) ? 4 : r_val );
			int str_idx = r_idx*5+l_idx;
			/*
			if( (str_idx < 0) || (str_idx >= 25) ){
				printf("BAD! [%d](%f:%d,%f:%d) -> (%d)",j,l_val,l_idx,r_val,r_idx,str_idx);
			}
			*/
			printf("%s",lookup[str_idx]);
		}
		printf("\n");
	}

	int   rule_size = 8*width/2;
	char* rule_vals = new char[rule_size]; 
	memset(rule_vals,'\0',rule_size);

	printf("        ");
	for(int j=0; j<width; j+=2){
		float l_limit = low + ((high-low)/width)*j;
		sprintf(&rule_vals[(8*j/2)],"%7.3f",l_limit);
		printf("\\ ");
	}
	printf("\n");
	for(int i=0; i<7; i++){
		printf("        ");
		for(int j=0; j<width; j+=2){
			printf(" %c",rule_vals[(8*j/2)+i]);
		}
		printf("\n");
	}

	free(rule_vals);

}


















}




