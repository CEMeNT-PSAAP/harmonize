#include "hip/hip_runtime.h"


#define checkError  util::check_error


#include "fiss_common.cu"


struct GlobalState{
	unsigned int  start;
	unsigned int  limit;
	sim_params    params;
};

struct GroupState { util::GroupWorkIter<unsigned int> iterator; };

typedef ProgramStateDef<GlobalState,GroupState,VoidState> ProgState;

enum class Fn { Neutron };

DEF_PROMISE_TYPE(Fn::Neutron, unsigned int);

typedef  HarmonizeProgram < PromiseUnion<Fn::Neutron>, ProgState > ProgType;


DEF_ASYNC_FN(ProgType, Fn::Neutron, arg) {


	neutron n;
	#ifdef INDIRECT
	n = global.params.old_data[arg];
	#else
	n = arg;
	#endif

	#ifndef PRE_INIT
	if ( n.time <= 0 ){
		n.p_x = 0;
		n.p_y = 0;
		n.p_z = 0;
		n.time = 0.0;
		random_3D_iso_mom(n);
	}
	#endif


	for(int i=0; i < global.params.horizon; i++){
		if( ! step_neutron(global.params,n) ){
			return;
		}
	}

	#ifdef INDIRECT
	global.params.old_data[arg] = n;
	ASYNC_CALL(Fn::Neutron,arg);		
	#else
	ASYNC_CALL(Fn::Neutron,n);		
	#endif	



}


DEF_INITIALIZE(ProgType) {

	unsigned int group_data_size = (global.limit - global.start) / gridDim.x;
	unsigned int group_start = global.start + group_data_size * blockIdx.x;
	unsigned int group_end   = group_start + group_data_size;
	if( blockIdx.x == (gridDim.x-1) ){
		group_end = global.limit;
	}

	group.iterator.reset(group_start,group_end);


}


DEF_FINALIZE(ProgType) {


}


DEF_MAKE_WORK(ProgType) {


	unsigned int index;

	#if 1
	util::BasicIter<unsigned int> iter = group.iterator.multi_step<14>();

	while(iter.step(index)){
		#ifdef INDIRECT
		neutron n;
		n.seed   = index;

		#ifdef PRE_INIT
		n.p_x = 0.0;
		n.p_y = 0.0;
		n.p_z = 0.0;
		random_3D_iso_mom(n);
		n.time = 0.0;
		#else
		n.time   = -1.0;
		#endif //PRE_INIT

		global.params.old_data[index] = n;
		#else
		thunk.data[6] = (unsigned int) __float_as_uint(-1.0);
		thunk.data[7] = (unsigned int) id;	
		#endif //INDIRECT
		
		ASYNC_CALL(Fn::Neutron,index);
	}
	#else

	group.iterator.step(index);

	#ifdef INDIRECT
	neutron n;
	n.seed   = index;

	#ifdef PRE_INIT
	n.p_x = 0.0;
	n.p_y = 0.0;
	n.p_z = 0.0;
	random_3D_iso_mom(n);
	n.time = 0.0;
	#else
	n.time   = -1.0;
	#endif //PRE_INIT

	global.params.old_data[index] = n;
	#else
	thunk.data[6] = (unsigned int) __float_as_uint(-1.0);
	thunk.data[7] = (unsigned int) id;	
	#endif //INDIRECT
	
	ASYNC_CALL(Fn::Neutron,index);


	#endif

	return !group.iterator.done();

}



int main(int argc, char *argv[]){


	util::ArgSet args(argc,argv);

	common_context com;

	com = common_initialize(args);
	hipDeviceSynchronize();
		
	checkError();

	GlobalState gs;
	gs.start  = 0;
	gs.limit  = com.params.count_lim;
	gs.params = com.params;
	
	//printf("Making an instance...\n");
	ProgType::Instance instance = ProgType::Instance(0xFFFFF,gs);
	hipDeviceSynchronize();
	util::check_error();
	
	//printf("Initing an instance...\n");
	init<ProgType>(instance,DEF_WG_COUNT);
	hipDeviceSynchronize();
	util::check_error();

	//printf("Execing an instance...\n");
	exec<ProgType>(instance,DEF_WG_COUNT,0xFFFFF);
	hipDeviceSynchronize();
	util::check_error();
	//printf("Finished exec.\n");
	

	common_finalize(com);

	return 0;

}

