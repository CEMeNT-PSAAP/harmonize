#include "hip/hip_runtime.h"
#include "fiss_common.cu"


using namespace util;





__global__ void sim_pass(SimParams params){


	while( ! params.source_id_iter->done() ) {

		Iter<unsigned int> iter = params.source_id_iter->leap(1);
		
		unsigned int id;
		while ( iter.step(id) ){

			Neutron n(id,0.0,0.0,0.0,0.0);

			int result = 0;		
			while ( result == 0 ){
				result = step_neutron(params,n);			
			}

			if ( result == -1 ){
				break;
			}

			#ifdef IOBUFF
			for(int i=0; i<result; i++){
				Neutron new_neu(n);
				param.neutron_io.push(new_neu);
			}
			#endif
		
		}

	}

}



int main(int argc, char *argv[]){

	ArgSet args(argc,argv);

	unsigned int wg_count = args["wg_count"];
	unsigned int wg_size  = args["wg_size"] | 32u;

	CommonContext context(args); 
        hipDeviceSynchronize( );
	check_error();

	#ifdef IOBUFF
	sim_pass<<<wg_count,wg_size>>>(context.params);
	#else
	sim_pass<<<wg_count,wg_size>>>(context.params);
	#endif

	return 0;

}



