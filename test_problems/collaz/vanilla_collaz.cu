
#include <hip/hip_runtime.h>
#include <stdio.h>


#ifndef DEF_SPAN
	#define DEF_SPAN 65536
#endif



#ifndef DEF_MULTIPROCESSOR_COUNT
	#define DEF_MULTIPROCESSOR_COUNT 	1
#endif

#ifndef DEF_WARP_SIZE
	#define DEF_WARP_SIZE 			32
#endif

#ifndef DEF_FUNCTION_ID_COUNT
	#define DEF_FUNCTION_ID_COUNT 		4
#endif

#ifndef DEF_THUNK_SIZE
	#define DEF_THUNK_SIZE 			4
#endif

#ifndef DEF_STACK_MODE
	#define DEF_STACK_MODE			0
#endif

#ifndef DEF_RETRY_LIMIT
	#define DEF_RETRY_LIMIT			8
#endif



/*
// This helps in determining how many warps and threads can reasonably be running simultaneously.
*/
const unsigned int MULTIPROCESSOR_COUNT	= DEF_MULTIPROCESSOR_COUNT;
const unsigned int WARP_SIZE		= DEF_WARP_SIZE;
const unsigned int TEAMS_PER_SM = 1u;
const unsigned int TEAM_COUNT = TEAMS_PER_SM * MULTIPROCESSOR_COUNT;


void checkError(){

	hipError_t status = hipGetLastError();

	if(status != hipSuccess){
		const char* err_str = hipGetErrorString(status);
		printf("ERROR: \"%s\"\n",err_str);
	}

}



__device__ unsigned int *step_counts;

__global__ void basic_collaz(unsigned int start, unsigned int end){

	for(unsigned int offset=blockIdx.x*WARP_SIZE+start; offset < end; offset+= TEAM_COUNT*WARP_SIZE){

		unsigned int original = offset+threadIdx.x;
		unsigned long long int val = original;
		unsigned int steps = 0;
		while(val > 1){
			if( (val % 2) == 0 ){
				val = val / 2;
			} else {
				val = val * 3 + 1;
			}
			steps++;
		}
		step_counts[original] = steps;
	}

}



unsigned int collaz(unsigned long long int val){

	unsigned int result = 0;
	while(val > 1){
		if( (val % 2) == 0 ){
			val = val / 2;
		} else {
			val = val * 3 + 1;
		}
		result++;
	}
	return result;

}



struct program_context{

	unsigned int* step_counts_ptr;
	hipEvent_t start;
	hipEvent_t stop;

};



const unsigned int span = DEF_SPAN;



program_context* initialize(){

	program_context* result = new program_context;

        hipEventCreate( &result->start );
        hipEventCreate( &result->stop  );
	hipDeviceSynchronize();
	hipEventRecord( result->start, NULL );


	hipMalloc( (void**) &result->step_counts_ptr, sizeof(unsigned int)*span );	
	checkError();
	
	hipError_t succ = hipMemcpyToSymbol(HIP_SYMBOL(step_counts),&result->step_counts_ptr,sizeof(unsigned int*));
	if(succ == hipSuccess){
		//printf("\n\nInitialized step_counts\n\n\n");
	} else if(succ == hipErrorInvalidSymbol){
		printf("\n\nInvalid symbol!\n\n\n");
	} else {
		printf("\n\nUh Oh\n\n\n");
	}

        hipDeviceSynchronize( );

	checkError();
	//printf("About to do the thing\n");
	
	return result;

}



void finalize(program_context* program){

	checkError();
	hipEventRecord( program->stop, NULL );

        // wait for the stop event to complete:
        hipDeviceSynchronize( );
	
	//printf("Just did the thing.\n");
	checkError();
	
	hipEventSynchronize( program->stop );

        float msecTotal = 0.0f;
        hipEventElapsedTime( &msecTotal, program->start, program->stop );
	
	
	

	unsigned int* host_results = (unsigned int*) malloc(sizeof(unsigned int)*span);
	hipMemcpy((void*)host_results,program->step_counts_ptr,sizeof(unsigned int)*span,hipMemcpyDeviceToHost);

	bool any_failure = false;
	unsigned int avg = 0;
	for(unsigned int i=0; i<span; i++){
		//printf("%d\t:\t%d",i,host_results[i]);
		unsigned int val = collaz(i);
		avg += val;
		if( host_results[i] == val ){
			//printf("\tS\n");
			;
		} else {
			any_failure = true;
			//printf("\tF\n");
		}
	}
	if(any_failure){
		printf("Failure encountered\n");
	} else {
		//printf("No failure found\n");
	}
	avg /= span;
	//printf("Average iteration value is %d\n",avg);
	//printf("\nSpan: %d\tTEAM_COUNT: %d\tWARP_SIZE: %d\n\n",span,TEAM_COUNT,WARP_SIZE);
	printf("%f",msecTotal);

}


int main(){

	program_context* program = initialize();	
	basic_collaz<<<TEAM_COUNT,WARP_SIZE>>>(0,span);
	finalize(program);

	return 0;
}

